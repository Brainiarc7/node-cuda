
#include <hip/hip_runtime.h>
#include <cstdio>

extern "C" {
  __global__ void helloWorld() {
    #if __CUDA_ARCH__ >= 200
    printf("Hello, world! I'm thread (%d,%d,%d) in block (%d,%d,%d).\n",
      threadIdx.x, threadIdx.y, threadIdx.z,
      blockIdx.x, blockIdx.y, blockIdx.z);
    #endif
  }
}
